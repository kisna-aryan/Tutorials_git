#include "hip/hip_runtime.h"
#include<iostream>
#include<opencv2/highgui/highgui.hpp>
#include<opencv2/imgproc/imgproc.hpp>
 
using namespace std;
using namespace cv;

#define NoOfBins 65536
 
int main()
{
    // Mat image = imread("nature.jpg", 0);
    Mat image = imread("images/frameIndex_0.png", -1);

    if( image.empty() )
    {
        cout << "Image not Found" << endl;
        return EXIT_FAILURE;
    }
    // allcoate memory for no of pixels for each intensity value
    /*     The maximum number of pixels can be total number of pixels in image.

    Total number of pixels in image resolution 640x512 is = 327680 

    The number of bins in 16 bit image is => 2^16 = 65536
    */
    int histogram[NoOfBins];
 
    // initialize all intensity values to 0
    for(int i = 0; i < NoOfBins; i++)
    {
        histogram[i] = 0;
    }
    
    // cout << sizeof(unsigned short)<< endl;
    cout << "pixel value:" << image.at<u_int16_t>(100,100)<< endl;
    cout <<" Channels:" << image.channels()<< endl;

    // calculate the no of pixels for each intensity values
    for(int y = 0; y < image.rows; y++)
    {
        for(int x = 0; x < image.cols; x++)
        {
            histogram[(int)image.at<u_int16_t>(y,x)]++;
        }
    }

 
/*     for(int i = 0; i < NoOfBins; i++)
        cout<<histogram[i]<<" "; */
 
    // draw the histograms
    int hist_w = 512; int hist_h = 400;
    // int bin_w = cvRound((double) hist_w/NoOfBins);
    double bin_w = (double) hist_w/NoOfBins;

 
    Mat histImage(hist_h, hist_w, CV_8UC1, Scalar(255, 255, 255));
 
    // find the maximum intensity element from histogram
    int max = histogram[0];
    for(int i = 1; i < NoOfBins; i++){
        if(max < histogram[i]){
            max = histogram[i];
        }
    }
 
    // normalize the histogram between 0 and histImage.rows
 
    for(int i = 0; i < NoOfBins; i++){
        histogram[i] = ((double)histogram[i]/max)*histImage.rows;
    }
 
 
    // draw the intensity line for histogram
    for(int i = 0; i < NoOfBins; i++)
    {
        line(histImage, Point(cvRound(bin_w*(i)), hist_h),
                              Point(cvRound(bin_w*(i)), hist_h - histogram[i]),
             Scalar(0,0,0), 1, 8, 0);
    }
 
    // display histogram
    namedWindow("Intensity Histogram");
    imshow("Intensity Histogram", histImage);
 
    namedWindow("Image");
    imshow("Image", image);
    waitKey();
    return 0;
}